#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include "time.h"

#define SPECTRA     512
#define CHANNELS    16384
#define SAMPLES     CHANNELS * SPECTRA

void main()
{
    // Normal mode
    // data buffer on the host
    hipfftReal *data_host = (hipfftReal*) malloc(SAMPLES * sizeof(hipfftReal));   
    hipfftComplex *data_host_out = (hipfftComplex*) malloc(SAMPLES * sizeof(hipfftComplex));
    // data buffer on the GPU
    hipfftComplex *data_gpu_out;
    hipfftReal *data_gpu;
    hipMalloc((void**)&data_gpu, SAMPLES * sizeof(hipfftReal));
    hipMalloc((void**)&data_gpu_out, SAMPLES * sizeof(hipfftComplex));
    // copy data from host to GPU
     hipMemcpy(data_gpu, data_host, SAMPLES * sizeof(hipfftReal), hipMemcpyHostToDevice);
    // cufft
    hipfftPlanMany(...);
    hipfftExecR2C(...);
    // copy result from GPU to host
    hipMemcpy(data_host_out, data_gpu_out, SAMPLES * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    // Zero copy mode
    // data buffer on the host
    hipfftComplex *data_host_out;
    hipHostAlloc((void **)&data_host_out, SAMPLES * sizeof(hipfftComplex), hipHostMallocMapped);
    hipfftReal *data_host;
    hipHostAlloc((void **)&data_host, SAMPLES * sizeof(hipfftReal), hipHostMallocMapped);
    // share the memory between host and GPU
    hipfftComplex *data_gpu_out;
    hipfftReal *data_gpu;
    hipHostGetDevicePointer((void**)&data_gpu, data_host, 0);
    hipHostGetDevicePointer((void**)&data_gpu_out, data_host_out, 0);
    // cufft
    hipfftPlanMany(...);
    hipfftExecR2C(...);
    
}

