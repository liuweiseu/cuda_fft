#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include "time.h"

#include "pfb_fir.cuh"

#define WGS         128
#define CHANNELS    16384
#define TAPS        4
#define SPECTRA     512
#define SAMPLES     CHANNELS * (SPECTRA + TAPS - 1)

#define WR_TO_FILE
#define NORMAL

#define REPEAT      6
#define ELAPSED_NS(start,stop) \
  (((int64_t)stop.tv_sec-start.tv_sec)*1000*1000*1000+(stop.tv_nsec-start.tv_nsec))

void gen_fake_data(float *data) {
   float fs = 1024;
   float fin  = 128;
   for( size_t t=0; t<SAMPLES; t++ ) { 
       double f = 2*M_PI * t *fin/fs;
       float res = 127 * sin(f)+127;
       *(data+t) = res;
       //*(data+t) = 1;
   }
}

int GetDevInfo()
{
    hipDeviceProp_t prop;
    int deviceID;
    hipGetDevice(&deviceID);
    hipGetDeviceProperties(&prop, deviceID);
    printf("GPU Device Info:\r\n");
    printf("%-25s: %d\r\n", "MaxThreadsPerBlock", prop.maxThreadsPerBlock);
    printf("%-25s: %d %d %d\r\n","maxThreadsDim", prop.maxThreadsDim[0], \
                                                  prop.maxThreadsDim[1], \
                                                  prop.maxThreadsDim[2]);
    printf("%-25s: %d %d %d\r\n","maxGridSize", prop.maxGridSize[0], \
                                                prop.maxGridSize[1], \
                                                prop.maxGridSize[2]);

    if(!prop.deviceOverlap)
        return -1;
    else
        return 0;
}

int main()
{
    struct timespec start, stop;
    int64_t elapsed_gpu_ns  = 0;

    hipfftReal test0 = 4;
    float test1 = 4;
    printf("%f %f\r\n",(float)test0,(hipfftReal)test1);
    printf("sizeof hipfftReal: %ld\r\n",sizeof(hipfftReal));

    int gpu_status = 0;
    gpu_status = GetDevInfo();
    if(gpu_status < 0)
        printf("No device will handle overlaps.\r\n");
    else   
        printf("overlaps are supported on the device.\r\n");

    //////////////////////////////////////////////////////////////////////////////////////////
    /*
    * preparing for pfb_fir
    */
    float *weights;
    weights = (float*) malloc(TAPS*CHANNELS*sizeof(float));
    printf("preparing for weights...\r\n");
    for(int i = 0; i<(TAPS*CHANNELS); i++)weights[i] = 1.0;
    printf("weights ready.\r\n");
    float *weights_gpu;
    hipMalloc((void**)&weights_gpu, TAPS*CHANNELS*sizeof(float));
    hipMemcpy(weights_gpu, weights, TAPS*CHANNELS*sizeof(float), hipMemcpyHostToDevice);

    hipfftReal *pfbfir_out_gpu;
    hipMalloc((void**)&pfbfir_out_gpu, CHANNELS*SPECTRA*sizeof(hipfftReal));

    float *pfbfir = (float*) malloc(sizeof(float)*CHANNELS*SPECTRA);
    for(int i = 0;i<CHANNELS*SPECTRA;i++)pfbfir[i] = 4.0;
    //hipMemcpy(pfbfir_out_gpu, pfbfir, CHANNELS*SPECTRA*sizeof(float), hipMemcpyHostToDevice);
    free(pfbfir);

    long long int step = CHANNELS;
    printf("%-10s : %lld\r\n","step",step);
    long long int out_n = step * SPECTRA;
    printf("%-10s : %lld\r\n","out_n",out_n);
    long long int stepy = (step * out_n + 256 * 1024 - 1)/(256*1024);
    printf("%-10s : %lld\r\n","stepy",stepy);
    int groupsx = step/WGS;
    printf("%-10s : %d\r\n","groupsx",groupsx);
    int groupsy = (out_n + stepy - 1)/stepy;
    printf("%-10s : %d\r\n","groupsy",groupsy);
    dim3 dimgrid(groupsx*WGS, groupsy);
    dim3 dimblock(WGS,1);
    ///////////////////////////////////////////////////////////////////////////////////////////

 #ifdef NORMAL
    printf("Normal Mode\r\n");
    // data buffer on the host computer
    //hipfftReal *data_host = (hipfftReal*) malloc(SAMPLES * sizeof(hipfftReal));
    unsigned char *data_host = (unsigned char*) malloc(SAMPLES * sizeof(unsigned char));   
    hipfftComplex *data_host_out = (hipfftComplex*) malloc(SAMPLES * sizeof(hipfftComplex));
 #else
    printf("Zero Copy Mode\r\n");
    hipfftComplex *data_host_out;
    hipHostAlloc((void **)&data_host_out, SAMPLES * sizeof(hipfftComplex), hipHostMallocMapped);
    hipfftReal *data_host;
    hipHostAlloc((void **)&data_host, SAMPLES * sizeof(hipfftReal), hipHostMallocMapped);
    //unsigned char *data_host;
    //hipHostAlloc((void **)&data_host, SAMPLES * sizeof(unsigned char), hipHostMallocMapped);
#endif

    int64_t elapsed_gpu_ns3  = 0;
    clock_gettime(CLOCK_MONOTONIC, &start);
    // generate fake data
    float *fake_data = (float*) malloc(SAMPLES * sizeof(float));
    gen_fake_data(fake_data);
    clock_gettime(CLOCK_MONOTONIC, &stop);
    elapsed_gpu_ns3 = ELAPSED_NS(start, stop);
    printf("%-25s: %f ms\r\n","Generating fake data time", elapsed_gpu_ns3/1000000.0);
    // init data buffer
    for(int i = 0; i < SAMPLES; i++)
    {
        data_host[i] = fake_data[i];
    }

    // data buffer on GPU
    hipfftComplex *data_gpu_out;
    //hipfftReal *data_gpu;
    unsigned char *data_gpu;
#ifdef NORMAL
    //hipMalloc((void**)&data_gpu, SAMPLES * sizeof(hipfftReal));
    hipMalloc((void**)&data_gpu, SAMPLES * sizeof(unsigned char));
    hipMalloc((void**)&data_gpu_out, SAMPLES * sizeof(hipfftComplex));
#else
    // do nothing here
#endif

    // exec fft
    hipfftHandle plan;
    /*
    *   1d fft
    */
    //hipfftPlan1d(&plan, SAMPLES, HIPFFT_C2C,1);
    
    /*
    * Many fft
    */
    int rank = 1;
    int n[1];
    n[0] = CHANNELS;
    int istride = 1;
    int idist = CHANNELS;
    int ostride = 1;
    int odist = CHANNELS;
    
    int inembed[1], onembed[1];
    inembed[0] = SAMPLES;
    onembed[0] = SAMPLES;
    //inembed[1] = SPECTRA;
    //onembed[1] = SPECTRA;
    
    hipfftResult fft_ret = hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, SPECTRA);
    //hipfftResult fft_ret = hipfftPlanMany(&plan, rank, n, NULL, istride, idist, NULL, ostride, odist, HIPFFT_C2C, SPECTRA);

    if( fft_ret != HIPFFT_SUCCESS ) {
        printf("hipfftPlanMany failed\r\n");
    }

    // record the start time
    int64_t elapsed_gpu_ns0  = 0;
    clock_gettime(CLOCK_MONOTONIC, &start);
    for(int i = 0; i < REPEAT; i++)
    {
    
    // copy data from host to GPU
#ifdef NORMAL
        //hipMemcpy(data_gpu, data_host, SAMPLES * sizeof(hipfftReal), hipMemcpyHostToDevice);
        hipMemcpy(data_gpu, data_host, SAMPLES * sizeof(unsigned char), hipMemcpyHostToDevice);
        //hipMemset(data_gpu,1, SAMPLES*sizeof(hipfftReal));
#else
        hipHostGetDevicePointer((void**)&data_gpu, data_host, 0);
        hipHostGetDevicePointer((void**)&data_gpu_out, data_host_out, 0);
#endif
        
        pfb_fir<<<dimgrid,dimblock>>>(
        (float *)pfbfir_out_gpu,  
        (unsigned char*)data_gpu,   
        weights_gpu,    
        out_n,
        step,
        stepy,
        0,
        0
        );
     

        //hipfftExecC2C(plan, (hipfftComplex*) data_gpu, (hipfftComplex*) data_gpu, HIPFFT_FORWARD);


        fft_ret = hipfftExecR2C(plan, (hipfftReal*)pfbfir_out_gpu, (hipfftComplex*) data_gpu_out);
        //fft_ret = hipfftExecR2C(plan, (hipfftReal*)data_gpu, (hipfftComplex*) data_gpu_out);
        if (fft_ret != HIPFFT_SUCCESS) {
            printf("forward transform fail\r\n"); 
        }
    }
    /*
    hipfftReal *pfbfir_out = (hipfftReal*)malloc(CHANNELS*SPECTRA*sizeof(hipfftReal));
    hipMemcpy(pfbfir_out, pfbfir_out_gpu, CHANNELS*SPECTRA*sizeof(hipfftReal), hipMemcpyDeviceToHost);
    for(long long unsigned int i = 0;i<CHANNELS*SPECTRA ;i++)
    {
        if(pfbfir_out[i]!=4.0)
        {
            printf("%lld %f\r\n",i,pfbfir_out[i]);
            break;
        }       
    }
    for(int i = 0;i<200;i++)printf("%d %f\r\n",i,pfbfir_out[i]);
    free(pfbfir_out);
    */
    hipDeviceSynchronize();
    // record the end time
    clock_gettime(CLOCK_MONOTONIC, &stop);
    elapsed_gpu_ns0 = ELAPSED_NS(start, stop);
    printf("%-25s: %f ms\r\n","Processing and copy time", elapsed_gpu_ns0/1000000.0);

    // copy data from GPU to host
    int64_t elapsed_gpu_ns2  = 0;
    clock_gettime(CLOCK_MONOTONIC, &start);
#ifdef NORMAL
    hipMemcpy(data_host_out, data_gpu_out, SAMPLES * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
#else
    // do nothing here
#endif
    clock_gettime(CLOCK_MONOTONIC, &stop);
    elapsed_gpu_ns2 = ELAPSED_NS(start, stop);
    printf("%-25s: %f ms\r\n","copy time(dev to host)", elapsed_gpu_ns2/1000000.0);

    elapsed_gpu_ns = elapsed_gpu_ns0  + elapsed_gpu_ns2;
    printf("%-25s: %f ms\r\n","total time", elapsed_gpu_ns/1000000.0);

    // cal power
    float *res = (float*) malloc(SAMPLES * sizeof(float));
    for(int i = 0; i < SAMPLES; i++)
    {
        res[i] = data_host_out[i].x * data_host_out[i].x + data_host_out[i].y * data_host_out[i].y;
    }

    // write data to file
#ifdef WR_TO_FILE
    FILE *fp;
    fp = fopen("fft.dat","w");
    if(fp==NULL)
    {
        fprintf(stderr, "the file can not be create.\r\n");
        return -1;
    }
    else
    {
        fprintf(stderr, "file created.\r\n");
    }
    fwrite(res,SAMPLES*sizeof(float),1,fp);
    fclose(fp);
#else
    // do nothing
#endif
    
    // end
    hipFree(weights_gpu);
    hipFree(pfbfir_out_gpu);
    hipfftDestroy(plan);
    hipFree(data_gpu_out);
    free(res);
#ifdef NORMAL
    hipFree(data_gpu);
    free(data_host);
#else
    hipHostFree(data_host);
#endif

    return 0;
}