/*******************************************************************************
All of the GPU related code are here.
We will compile the code as a .so, and then link the code in the hashpipe code.
 ******************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include "time.h"

#include "fast_gpu.h"
// This is the PFB FIR code from James
#include "pfb_fir.cuh"

// The following buffers are the GPU buffers.
char            *data_in_gpu;       // input data
float           *weights_gpu;       // PFB FIR weights
hipfftReal       *pfbfir_out_gpu;    // the output of PFB FIR
hipfftComplex    *data_out_gpu;      // output data on GPU
hipfftComplex    *data_out_host;     // output data on host

// cufft plan
hipfftHandle plan;

int GPU_GetDevInfo()
{
    hipDeviceProp_t prop;
    int deviceID;
    hipGetDevice(&deviceID);
    hipGetDeviceProperties(&prop, deviceID);
    printf("GPU Device Info:\r\n");
    printf("%-25s: %d\r\n", "MaxThreadsPerBlock", prop.maxThreadsPerBlock);
    printf("%-25s: %d %d %d\r\n","maxThreadsDim", prop.maxThreadsDim[0], \
                                                  prop.maxThreadsDim[1], \
                                                  prop.maxThreadsDim[2]);
    printf("%-25s: %d %d %d\r\n","maxGridSize",   prop.maxGridSize[0], \
                                                  prop.maxGridSize[1], \
                                                  prop.maxGridSize[2]);

    if(!prop.deviceOverlap)
        return -1;
    else
        return 0;
}

// This func is used for allocating pinned memory on the host computer 
int Host_BufferInit(DIN_TYPE *buf_in, DOUT_TYPE *buf_out)
{
    hipError_t status;
    status = hipHostMalloc((void **)&buf_in,SAMPLES * sizeof(DIN_TYPE));
    if(status != hipSuccess)
        return -1;
    status = hipHostMalloc((void **)&buf_out, OUTPUT_LEN * sizeof(float));
    if(status != hipSuccess)
        return -2;
    return 0;
}

// This func is used for allocating memory on the GPU
void GPU_BufferInit()
{
    hipMalloc((void**)&data_in_gpu, SAMPLES * sizeof(char));
    hipMalloc((void**)&weights_gpu, TAPS*CHANNELS*sizeof(float));
    hipMalloc((void**)&pfbfir_out_gpu, CHANNELS*SPECTRA*sizeof(hipfftReal));
    hipMalloc((void**)&data_out_gpu, CHANNELS*SPECTRA * sizeof(hipfftComplex));
}

// This func is used for creating cufft plan
int GPU_CreateFFTPlan()
{
    int rank = 1;
    int n[1];
    n[0] = CHANNELS;
    int istride = 1;
    int idist = CHANNELS;
    int ostride = 1;
    int odist = CHANNELS;
    
    int inembed[1], onembed[1];
    inembed[0] = CHANNELS*SPECTRA;
    onembed[0] = CHANNELS*SPECTRA;
    hipfftResult fft_ret = hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, SPECTRA);
    if( fft_ret != HIPFFT_SUCCESS )
        return -1;
    else
        return 0;
}

// move data from host to GPU
void GPU_MoveDataFromHost(DIN_TYPE *din)
{
    hipMemcpy(data_in_gpu, din, SAMPLES * sizeof(DIN_TYPE), hipMemcpyHostToDevice);
}

// move data from GPU to host
void GPU_MoveDataToHost(DOUT_TYPE *dout)
{
    hipMemcpy(data_host_out, data_gpu_out, CHANNELS*SPECTRA * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
}

// calculate power of the output data
void CalPower(DOUT_TYPE *res)
{

}